#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

// Lab 9 : Rahil Sharma
//compile : nvcc -o fractal_par fractal_par.cu -lpthread -lglut -lGL
//run : ./fractal_par
//(also)run : double click on the object file created after compilation 
#include "book.h"
#include "cpu_bitmap.h"

#define DIM 500

 struct hipComplex {
 float r;
 float i;
 __device__ hipComplex( float a, float b ) : r(a), i(b) {}
 __device__ float magnitude2( void ) {
 return r * r + i * i;
 }
 __device__ hipComplex operator*(const hipComplex& a) {
 return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
 }
 __device__ hipComplex operator+(const hipComplex& a) {
 return hipComplex(r+a.r, i+a.i);
 }
};


__device__ int julia( int x, int y ) {
 const float scale = 1.5;
 float jx = scale * (float)(DIM/2 - x)/(DIM/2);
 float jy = scale * (float)(DIM/2 - y)/(DIM/2);
 hipComplex c(-0.8, 0.156);
 hipComplex a(jx, jy);
 int i = 0;
 for (i=0; i<200; i++) {
 a = a * a + c;
 if (a.magnitude2() > 1000)
 return 0;
 }
 return 1;
}

__global__ void kernel( unsigned char *ptr ) {
 int x = blockIdx.x;
 int y = blockIdx.y;
 int offset = x + y * gridDim.x;
 int juliaValue = julia( x, y );
    ptr[offset*4 + 0] = 0;
    ptr[offset*4 + 1] = 255 *juliaValue;
    ptr[offset*4 + 2] = ((x*y)%255);
    ptr[offset*4 + 3] = 0;
}
int main( void ) {
 CPUBitmap bitmap( DIM, DIM );
 unsigned char *dev_bitmap;
 HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, 
 bitmap.image_size() ) );
 dim3 grid(DIM,DIM);
 kernel<<<grid,1>>>( dev_bitmap );
 HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost ) );
 bitmap.display_and_exit(NULL);
 HANDLE_ERROR( hipFree( dev_bitmap ) );
}
